#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// monte carlo simulation (CUDA Kernel) on the device

__global__  void MonteCarlo( float *X, float *Y, float *R , int *hits)
{
  // calculate global id
  /* unsigned int numItems = blockDim.x; */
  /* unsigned int tnum = threadIdx.x; */
  /* unsigned int wgNum = blockIdx.x; */
  unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

  // randomize the location and radius of the circle:
  float xc = X[gid];
  float yc = Y[gid];
  float  r = R[gid];

  // solve for the intersection using the quadratic formula:
  float a = 2.;
  float b = -2.*( xc + yc );
  float c = xc*xc + yc*yc - r*r;
  float d = b*b - 4.*a*c;

  // If d is less than 0., then the circle was completely missed. (Case A) Ignore this case
  if (d >= 0) {

    // get the first intersection:
    d = sqrt( d );
    float t1 = (-b + d ) / ( 2.*a );	// time to intersect the circle
    float t2 = (-b - d ) / ( 2.*a );	// time to intersect the circle
    float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection
    
    // If tmin is less than 0., then the circle completely engulfs the laser pointer. (Case B) Ignore this case
    if (tmin >= 0.) {
      // where does it intersect the circle?
      float xcir = tmin;
      float ycir = tmin;

      // get the unitized normal vector at the point of intersection:
      float nx = xcir - xc;
      float ny = ycir - yc;
      float n = sqrt( nx*nx + ny*ny );
      nx /= n;	// unit vector
      ny /= n;	// unit vector

      // get the unitized incoming vector:
      float inx = xcir - 0.;
      float iny = ycir - 0.;
      float in = sqrt( inx*inx + iny*iny );
      inx /= in;	// unit vector
      iny /= in;	// unit vector

      // get the outgoing (bounced) vector:
      float dot = inx*nx + iny*ny;
      /* float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence` */
      float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

      // find out if it hits the infinite plate:
      float t = ( 0. - ycir ) / outy;

      // If t is less than 0., then the reflected beam went up instead of down. Ignore this case
      // Otherwise, this beam hit the infinite plate. (Case D) Set hit = 1 for this case
      if (t >= 0) {
        hits[gid] = 1;
      }
    }
  }
}

/* // ranges for the random numbers: */
const float XCMIN =	 0.0;
const float XCMAX =	 2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;

// function prototypes:
float		Ranf( float, float );
int		Ranf( int, int );
void		TimeOfDaySeed( );

int
main( int argc, char* argv[ ] )
{

  TimeOfDaySeed( );		// seed the random number generator

  // arrays on host to hold random values for x/y coords, radius, and hits
  float *hX = new float [SIZE];                 // x coords
  float *hY = new float [SIZE];                 // y coords
  float *hR = new float [SIZE];                 // radius coords
  int   *hH = new int   [SIZE];                 // holds hits from each block

  // fill the random-value arrays:
  for( int n = 0; n < SIZE; n++ )
  {       
    hX[n] = Ranf( XCMIN, XCMAX );
    hY[n] = Ranf( YCMIN, YCMAX );
    hR[n] = Ranf(  RMIN,  RMAX ); 
    hH[n] = 0; 
  }       

  /* int dev = findCudaDevice(argc, (const char **)argv); */

  // allocate device memory:

  float *dX, *dY, *dR;
  int *dH;

  dim3 dimsX( SIZE, 1, 1 );
  dim3 dimsY( SIZE, 1, 1 );
  dim3 dimsR( SIZE, 1, 1 );
  dim3 dimsH( SIZE, 1, 1 );

  hipError_t status;
  status = hipMalloc( reinterpret_cast<void **>(&dX), SIZE*sizeof(float) );
          checkCudaErrors( status );
  status = hipMalloc( reinterpret_cast<void **>(&dY), SIZE*sizeof(float) );
          checkCudaErrors( status );
  status = hipMalloc( reinterpret_cast<void **>(&dR), SIZE*sizeof(float) );
          checkCudaErrors( status );
  status = hipMalloc( reinterpret_cast<void **>(&dH), (SIZE)*sizeof(int) );
          checkCudaErrors( status );

  // copy host memory to the device:
  status = hipMemcpy( dX, hX, SIZE*sizeof(float), hipMemcpyHostToDevice );
          checkCudaErrors( status );
  status = hipMemcpy( dY, hY, SIZE*sizeof(float), hipMemcpyHostToDevice );
          checkCudaErrors( status );
  status = hipMemcpy( dR, hR, SIZE*sizeof(float), hipMemcpyHostToDevice );
          checkCudaErrors( status );
  status = hipMemcpy( dH, hH, SIZE*sizeof(int), hipMemcpyHostToDevice );
          checkCudaErrors( status );

  // setup the execution parameters:

  dim3 threads(BLOCKSIZE, 1, 1 );
  dim3 grid( SIZE / threads.x, 1, 1 );

  // Create and start timer

  hipDeviceSynchronize( );

  // allocate CUDA events that we'll use for timing:

  hipEvent_t start, stop;
  status = hipEventCreate( &start );
          checkCudaErrors( status );
  status = hipEventCreate( &stop );
          checkCudaErrors( status );

  // record the start event:

  status = hipEventRecord( start, NULL );
  checkCudaErrors( status );

  // execute the kernel:
  for( int t = 0; t < NUMTRIALS; t++)
  {
    MonteCarlo<<< grid, threads >>>( dX, dY, dR, dH );
  }

  // record the stop event:

  status = hipEventRecord( stop, NULL );
  checkCudaErrors( status );

  // wait for the stop event to complete:

  status = hipEventSynchronize( stop );
  checkCudaErrors( status );

  float msecTotal = 0.0f;
  status = hipEventElapsedTime( &msecTotal, start, stop );
  checkCudaErrors( status );

  // compute and print the performance

  double secondsTotal = 0.001 * (double)msecTotal;
  double trialsPerSecond = (double)NUMTRIALS * (double)SIZE / secondsTotal;
  double megaTrialsPerSecond = trialsPerSecond / 1000000.;

  // copy result from the device to the host:
  status = hipMemcpy( hH, dH, (SIZE)*sizeof(int), hipMemcpyDeviceToHost );
  checkCudaErrors( status );

  // calculate probability (only uses results from last iteration of NUMTRIALS
  int numHits = 0;
  for (int i = 0; i < SIZE; i++) 
  {
    numHits += hH[i];
  }
  double probability = (double) numHits / (double) SIZE;

  printf("%10d\t%10d\t%10.2lf\t%10.2lf\n", BLOCKSIZE, SIZE, megaTrialsPerSecond, probability );

  // clean up memory:
  delete [ ] hX;
  delete [ ] hY;
  delete [ ] hR;
  delete [ ] hH;

  status = hipFree( dX );
  checkCudaErrors( status );
  status = hipFree( dY );
  checkCudaErrors( status );
  status = hipFree( dR );
  checkCudaErrors( status );
  status = hipFree( dH );
  checkCudaErrors( status );

  return 0;
}

float
Ranf( float low, float high )
{
  float r = (float) rand();               // 0 - RAND_MAX
  float t = r  /  (float) RAND_MAX;       // 0. - 1.

  return   low  +  t * ( high - low );
}

int
Ranf( int ilow, int ihigh )
{
  float low = (float)ilow;
  float high = ceil( (float)ihigh );

  return (int) Ranf(low,high);
}

void
TimeOfDaySeed( )
{
  struct tm y2k = { 0 };
  y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
  y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

  time_t  timer;
  time( &timer );
  double seconds = difftime( timer, mktime(&y2k) );
  unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
  srand( seed );
}
